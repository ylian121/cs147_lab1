#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

int main (int argc, char *argv[])
{

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t mat_sz;
    unsigned matDim;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matDim = 1000;
    } else if (argc == 2) {
        matDim = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
      "\n    Usage: ./mat-add                # All matrices are 1000 x 1000"
      "\n    Usage: ./mat-add <m>            # All matrices are m x m"
      "\n");
        exit(0);
    }
   
    mat_sz = matDim*matDim;

    A_h = (float*) malloc( sizeof(float)*mat_sz );
    for (unsigned int i=0; i < mat_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    B_h = (float*) malloc( sizeof(float)*mat_sz );
    for (unsigned int i=0; i < mat_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    C_h = (float*) malloc( sizeof(float)*mat_sz );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", matDim, matDim,
        matDim, matDim, matDim, matDim);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);
	
    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel using standard mat-add interface -------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);
    basicMatAdd(matDim, A_d, B_d, C_d);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Unable to launch kernel");
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables to host ------------------------------------------
    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results...\n"); fflush(stdout);

    verify(A_h, B_h, C_h, matDim);


    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    /*************************************************************************/
    //INSERT CODE HERE
    
    /*************************************************************************/
    return 0;
}

