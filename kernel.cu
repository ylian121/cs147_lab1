#include "hip/hip_runtime.h"
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i<dim)
      C[i] = A[i] + B[i];
        
    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
	
	// Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid(ceil(n/256.0, 1, 1);
    dim3 DimBlock(256, 1, 1);
    matAdd<<<DimGrid, DimBlock>>>(A, B, C, dim);
	
    /*************************************************************************/

}

