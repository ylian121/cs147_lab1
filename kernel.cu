
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void matAdd(int dim, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (dim x dim) matrix
     *   where B is a (dim x dim) matrix
     *   where C is a (dim x dim) matrix
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    int i = (idy*dim + idx);
    if(i<dim)
      C[i] = A[i] + B[i];
        
    /*************************************************************************/

}

void basicMatAdd(int dim, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE

    /*************************************************************************/
	
	// Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((dim-1)/BLOCK_SIZE+1, (dim-1)/BLOCK_SIZE+1, 1);
    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    matAdd<<<DimGrid, DimBlock>>>(dim, A, B, C);
	
    /*************************************************************************/

}

